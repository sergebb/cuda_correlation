#include "hip/hip_runtime.h"
#include "cuda_compute.h"
#include <stdio.h>
 
#define BLOCK_SIZE 512

__global__ 
void gCorrelationCompute( float *dev_input, float *dev_output, int rows, int cols, size_t pitch) 
{
    // blockIdx.x; 
    // blockIdx.y;               
    // threadIdx.x; 
    int xCoord = blockIdx.x*blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    int yCoord = blockIdx.y;

    __shared__ float sumBuf[BLOCK_SIZE];
    sumBuf[tid] = 0;
    int i,j,p;
    unsigned int s;
    float val;
    float *t;
    if( xCoord<cols-1 && yCoord<rows){
        val = *((float*)((char*)dev_input + yCoord * pitch) + xCoord);  //Value in memory cell (xCoord,yCoord) in first line, const for every thread
        for( j=yCoord; j<rows; j++ ){
            for( i=0; i<cols; i++ ){
                p = (i+xCoord)%cols;                                            //Coordinate in shifted line(second line)
                t = (float*)((char*)dev_input + j * pitch) + p;            //Value in memory cell (p,yCoord) multiply to (xCoord,yCoord)
                sumBuf[tid] = *t * val;                                 //Compute per element multiplication

                for( s=1; s<blockDim.x; s*=2) {         //Sum reduction
                    if(tid % (2*s) == 0) {
                        sumBuf[tid] += sumBuf[tid + s];
                    }
                    __syncthreads();
                }
                // write result for this block to global mem
                if(tid == 0) {
                    t = (float*)((char*)dev_output + yCoord * pitch) + i; //memory cell (i,yCoord) where results of correlation is saved
                    atomicAdd(t, sumBuf[0]);      //Results is sum of I(x)I(x+I) averaged for all x, this block calculate 512 x'es, other added from other blocks (that is why +=) 
                    if(j!=yCoord){
                        t = (float*)((char*)dev_output + j * pitch) + i;    //Add result to j(first) line and yCoord(second) line, 1to2 and 2to1 correlation results 
                        atomicAdd(t, sumBuf[0]);                            //are the same, so we save some time by fixind j>=yCoord and do not calculate it again.
                    }
                }
            }
        }
    }
}
 
int CudaCorrelate( float** input_data, float** output_data, int rows, int cols)
{
    float *dev_input,*dev_output,*t;
    hipError_t err;
    size_t pitch;
    //Memory allocation for input and output arrays
    err = hipMallocPitch((void**)&dev_input, &pitch, sizeof(float)*cols, rows);  
    if( err != hipSuccess ){
        fprintf( stderr, "Error at %s:%i : %s\n", __FILE__, __LINE__, hipGetErrorString( err ) );
        exit( -1 );
    } 
    err = hipMallocPitch((void**)&dev_output, &pitch, sizeof(float)*cols, rows);  
    if( err != hipSuccess ){
        fprintf( stderr, "Error at %s:%i : %s\n", __FILE__, __LINE__, hipGetErrorString( err ) );
        exit( -1 );
    } 
    //Memory copying&initialisation for input data and error-check
    for( int i=0; i<rows; i++ ){
        t = (float*)((char*)dev_input + i * pitch);// + Column;
        hipMemcpy( t, input_data[i], pitch, hipMemcpyHostToDevice );
    }
    for( int i=0; i<rows; i++ ){
        t = (float*)((char*)dev_output + i * pitch);// + Column;
        hipMemset( t, 0, pitch);
    }
    err = hipGetLastError();
    if( err != hipSuccess ){
        fprintf( stderr, "Error at %s:%i : %s\n", __FILE__, __LINE__, hipGetErrorString( err ) );
        exit( -1 );
    }

    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    //Calculation    
    dim3 dimBlock( BLOCK_SIZE, 1 );
    dim3 dimGrid( cols/BLOCK_SIZE + ((cols%BLOCK_SIZE==0)?0:1), rows );
    printf("block(%d,%d) and grid(%d,%d), cols=%d, rows=%d\n",dimBlock.x,dimBlock.y,dimGrid.x,dimGrid.y,cols,rows);
    gCorrelationCompute<<<dimGrid,dimBlock>>>( dev_input, dev_output, rows, cols, pitch );
    // hello<<<dimGrid, dimBlock>>>(ad, bd);
    err = hipDeviceSynchronize();
    if( err != hipSuccess ){
        fprintf( stderr, "Error at %s:%i : %s\n", __FILE__, __LINE__, hipGetErrorString( err ) );
        exit( -1 );
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf ("Kernel execution time: %f ms\n", time);


    //Result memory copying back
    for( int i=0; i<rows; i++ ){
        t = (float*)((char*)dev_output + i * pitch);// + Column;
        hipMemcpy( output_data[i], t, cols*sizeof(float), hipMemcpyDeviceToHost );
    }
    err = hipGetLastError();
    if( err != hipSuccess ){
        fprintf( stderr, "Error at %s:%i : %s\n", __FILE__, __LINE__, hipGetErrorString( err ) );
        exit( -1 );
    }

    hipFree( dev_input );
    hipFree( dev_output );
    
    return EXIT_SUCCESS;
}