#include "hip/hip_runtime.h"
#include "cuda_compute.h"
#include <stdio.h>
#include "hip/hip_math_constants.h"
 
#define BLOCK_SIZE 512
#define MAX_WIDTH 2048

texture<float, 2 > inputDataTexRef;

#define CUDA_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__
void gComputePolarProjection(float *dev_output, size_t pitchPolar, int image_number,
                             int rows, int cols, int r_min, int r_max, int polar_angles,
                             float center_y, float center_x, float cval)
{
    unsigned int r = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int t = blockIdx.y * blockDim.y + threadIdx.y;
    
    float radius = r_min + r;
    float angle_step = 2*HIP_PI_F/polar_angles;
    float angle = -HIP_PI_F + t*angle_step ;
    float res;

    if(radius < r_max && t < polar_angles){
        float x = radius * cos(angle) + center_x + 0.5f;
        float y = radius * sin(angle) + center_y + 0.5f;
        if(x<0 || x>= cols || y<0 || y>= rows){
            res = cval;
        }else{
            res = tex2D(inputDataTexRef, x, y + image_number*rows);
        }
        *((float*)((char*)dev_output + r * pitchPolar) + t) = res;
    }
}


__global__ void gCorrelationComputeLine( float *dev_input, float *dev_output, int rows, int cols, size_t pitchInput) 
{
    int xCoord = blockIdx.x*blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    int yCoord = blockIdx.y;

    __shared__ float line_data[MAX_WIDTH];
    int i,pos;
    float val;
    float t;
    float sum = 0;
    if( xCoord<cols && yCoord<rows){
        for(i=0; i*BLOCK_SIZE < cols; i++){
            if(tid+i*BLOCK_SIZE < cols){
                line_data[tid + i*BLOCK_SIZE] = *((float*)((char*)dev_input + yCoord * pitchInput) + tid + i*BLOCK_SIZE); //Copy line of matrix to shared memory
            }
        }
        __syncthreads();

        for( i=0; i<cols; i++ ){        //Loop over whole line
            val = line_data[i];         //First multiplier
            pos = (i+xCoord)%cols;      //Coordinate in shifted line(second line)
            t = line_data[pos];         //Value in memory cell (pos,yCoord), second multiplier
            sum += t * val;             //Sum of multiplications
        }
        sum /= cols;    //Divide by line len -> correlation
        
        *((float*)((char*)dev_output + yCoord * pitchInput) + xCoord) = sum; //memory cell (xCoord,yCoord) where results of correlation is saved
    }
}
    
__global__ 
void gRecoverMask( float *dev_input, int rows, int cols, size_t pitchInput) 
{
    int xCoord = threadIdx.x;
    int yCoord = blockIdx.y;
    int i,s;
    float val;
    float average_val;    
    
    float sum_value = 0;
    int non_mask = 0;
    __shared__ float sum_buf[BLOCK_SIZE];
    __shared__ float non_mask_buf[BLOCK_SIZE];

    for(i=0; i*BLOCK_SIZE < cols; i++){
        if(xCoord + i*BLOCK_SIZE < cols){
            val = *((float*)((char*)dev_input + yCoord * pitchInput) + xCoord + i*BLOCK_SIZE);
        }else{
            val = -1;
        }

        if(val >= 0){
            non_mask = 1;
        }else{
            val = 0;
            non_mask = 0;
        }
        
        sum_buf[xCoord] = val;
        non_mask_buf[xCoord] = non_mask;
        for( s=1; s<BLOCK_SIZE; s*=2) {         //Sum reduction
            if(xCoord % (2*s) == 0) {
                sum_buf[xCoord] += sum_buf[xCoord + s];
                non_mask_buf[xCoord] += non_mask_buf[xCoord + s];
            }
            __syncthreads();
        }
        if(xCoord == 0) {
            sum_value += sum_buf[0];
            non_mask += non_mask_buf[0];
        }
    }
    if(xCoord == 0) {
        sum_buf[0] = sum_value;
        non_mask_buf[0] = non_mask;
    }
    __syncthreads();
    average_val = sum_buf[0]/non_mask_buf[0];
    
    for(i=0; i*BLOCK_SIZE < cols; i++){
        if(xCoord + i*BLOCK_SIZE < cols){
            val = *((float*)((char*)dev_input + yCoord * pitchInput) + xCoord + i*BLOCK_SIZE);
            if(val < 0){
                *((float*)((char*)dev_input + yCoord * pitchInput) + xCoord + i*BLOCK_SIZE) = average_val;
            }
        }
    }
}
__global__
void gCCFAngle(float *dev_ccf_2d, float *dev_ccf_angle, int radius_range, size_t pitchF){
    int xCoord = blockIdx.x;
    int yCoord = threadIdx.y;
    int i,s;
    __shared__ float sum_buf[BLOCK_SIZE];
    float sum_value = 0;
    
    for(i=0; i*BLOCK_SIZE < radius_range; i++){
        if(yCoord + i*BLOCK_SIZE < radius_range){
            sum_buf[yCoord] = *((float*)((char*)dev_ccf_2d + (yCoord + i*BLOCK_SIZE) * pitchF) + xCoord);
        }else{
            sum_buf[yCoord] = 0;
        }
        for( s=1; s<BLOCK_SIZE; s*=2) {         //Sum reduction
            if(yCoord % (2*s) == 0) {
                sum_buf[yCoord] += sum_buf[yCoord + s];
            }
            __syncthreads();
        }
        if(yCoord == 0){
            sum_value += sum_buf[0];
        }
    }
    if(yCoord == 0){
        dev_ccf_angle[xCoord] = sum_value/radius_range;
    }
    
}


int CudaReprojectToPolar(float *input_data, size_t input_row_stride, float *polar_data, size_t polar_row_stride,
                         int rows, int cols, int r_min, int r_max, int polar_angles,
                         float center_y, float center_x, float cval)
{
    float *dev_input, *dev_output;
    size_t pitchInput, pitchPolar;
    
    // ///////////////////////
    // hipEvent_t start, stop;
    // float time;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    // hipEventRecord(start, 0);
    // ///////////////////////

    //Memory allocation for input and output arrays
    CUDA_CHECK(hipMallocPitch((void**)&dev_input, &pitchInput, sizeof(float)*cols, rows));
    CUDA_CHECK(hipMallocPitch((void**)&dev_output, &pitchPolar, sizeof(float)*polar_angles, r_max-r_min));

    // CUDA_CHECK(hipMemcpy(dev_input, input_data, rows*input_row_stride, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy2D(dev_input, pitchInput, input_data, input_row_stride, sizeof(float)*cols, rows, hipMemcpyHostToDevice));

    // Specify texture    
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    CUDA_CHECK(hipBindTexture2D(NULL, inputDataTexRef, dev_input, channelDesc, cols, rows, pitchInput));
    inputDataTexRef.addressMode[0] = hipAddressModeBorder;
    inputDataTexRef.addressMode[1] = hipAddressModeBorder;
    inputDataTexRef.filterMode = hipFilterModeLinear;
    inputDataTexRef.normalized = false;
    
    //Calculation    
    dim3 projBlock( 32, 32 );
    dim3 projGrid((r_max - r_min + projBlock.y - 1) / projBlock.y, 
                  (polar_angles + projBlock.x - 1) / projBlock.x);
    
    gComputePolarProjection<<<projGrid,projBlock>>>(dev_output, pitchPolar, 0,
                                                    rows, cols, r_min, r_max, polar_angles,
                                                    center_y, center_x, cval);
    
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy2D(polar_data, polar_row_stride, dev_output, pitchPolar, sizeof(float)*polar_angles, r_max-r_min, hipMemcpyDeviceToHost));

    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&time, start, stop);
    // printf ("CUDA execution time: %f ms\n", time);

    hipFree( dev_input );
    hipFree( dev_output );

    return EXIT_SUCCESS;
}

int CudaCorrelateLine(float* input_data, float* output_data, size_t numpy_row_stride, int rows, int cols)
{
    float *dev_input, *dev_output;
    size_t pitchInput;
    
    if( cols > MAX_WIDTH ){
        fprintf( stderr, "Error at %s:%i : %s\n", __FILE__, __LINE__, "Image width exceeds max value, need recompile" );
        exit( -1 );
    } 
    
    //Memory allocation for input and output arrays
    CUDA_CHECK(hipMallocPitch((void**)&dev_input, &pitchInput, sizeof(float)*cols, rows));
    CUDA_CHECK(hipMallocPitch((void**)&dev_output, &pitchInput, sizeof(float)*cols, rows));

    CUDA_CHECK(hipMemcpy2D(dev_input, pitchInput, input_data, numpy_row_stride, sizeof(float)*cols, rows, hipMemcpyHostToDevice));

    //RecoverMask
    dim3 recoverBlock( BLOCK_SIZE, 1 );
    dim3 recoverGrid( 1, rows );
    gRecoverMask<<<recoverGrid,recoverBlock>>>(dev_input, rows, cols, pitchInput);
    //Calculation
    dim3 corrBlock( BLOCK_SIZE, 1 );
    dim3 corrGrid( cols/BLOCK_SIZE + ((cols%BLOCK_SIZE==0)?0:1), rows );
    gCorrelationComputeLine<<<corrGrid,corrBlock>>>(dev_input, dev_output, rows, cols, pitchInput);
    
    CUDA_CHECK(hipDeviceSynchronize());
    
    CUDA_CHECK(hipMemcpy2D(output_data, numpy_row_stride, dev_output, pitchInput, sizeof(float)*cols, rows, hipMemcpyDeviceToHost));

    hipFree(dev_input);
    hipFree(dev_output);
    
    return EXIT_SUCCESS;
}

int CudaReprojectAndCorrelate(float* input_data, size_t input_row_stride, float* output_data, 
                              int rows, int cols, int r_min, int r_max, int polar_angles,
                              float center_y, float center_x, float cval)
{
    float *dev_input, *dev_polar_input, *dev_ccf_2d;
    float *dev_ccf_angle;
    size_t pitchInput, pitchPolar;
    
    // ///////////////////////
    // hipEvent_t start, stop;
    // float time;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    // hipEventRecord(start, 0);
    // ///////////////////////

    //Memory allocation for input and output arrays
    CUDA_CHECK(hipMallocPitch((void**)&dev_input, &pitchInput, sizeof(float)*cols, rows));
    CUDA_CHECK(hipMallocPitch((void**)&dev_polar_input, &pitchPolar, sizeof(float)*polar_angles, r_max-r_min));
    CUDA_CHECK(hipMallocPitch((void**)&dev_ccf_2d, &pitchPolar, sizeof(float)*polar_angles, r_max-r_min));
    CUDA_CHECK(hipMalloc((void**)&dev_ccf_angle, sizeof(float)*polar_angles));

    //Memory copying&initialisation for input data and error-check
    CUDA_CHECK(hipMemcpy2D(dev_input, pitchInput, input_data, input_row_stride, sizeof(float)*cols, rows, hipMemcpyHostToDevice));

    //Calculation    
    dim3 projBlock( 32, 32 );
    dim3 projGrid((r_max - r_min + projBlock.y - 1) / projBlock.y, 
                  (polar_angles + projBlock.x - 1) / projBlock.x);

    // Specify texture    
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    CUDA_CHECK(hipBindTexture2D(NULL, inputDataTexRef, dev_input, channelDesc, cols, rows, pitchInput));
    inputDataTexRef.addressMode[0] = hipAddressModeBorder;
    inputDataTexRef.addressMode[1] = hipAddressModeBorder;
    inputDataTexRef.filterMode = hipFilterModeLinear;
    inputDataTexRef.normalized = false;
    
    //Projection calculation
    gComputePolarProjection<<<projGrid,projBlock>>>(dev_polar_input, pitchPolar, 0,
                                                    rows, cols, r_min, r_max, polar_angles,
                                                    center_y, center_x, 0);

    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipUnbindTexture(inputDataTexRef));
    
    //RecoverMask
    dim3 recoverBlock( BLOCK_SIZE, 1 );
    dim3 recoverGrid( 1, r_max-r_min );
    gRecoverMask<<<recoverGrid,recoverBlock>>>( dev_polar_input, r_max-r_min, polar_angles, pitchPolar);

    dim3 corrBlock( BLOCK_SIZE, 1 );
    dim3 corrGrid( cols/BLOCK_SIZE + ((cols%BLOCK_SIZE==0)?0:1), rows );
    gCorrelationComputeLine<<<corrGrid,corrBlock>>>( dev_polar_input, dev_ccf_2d, r_max-r_min, polar_angles, pitchPolar);
    
    dim3 angleBlock( 1, BLOCK_SIZE );
    dim3 angleGrid( polar_angles, 1 );
    gCCFAngle<<<angleGrid,angleBlock>>>(dev_ccf_2d, dev_ccf_angle, r_max-r_min, pitchPolar);
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy( output_data, dev_ccf_angle, polar_angles*sizeof(float), hipMemcpyDeviceToHost ));

    // ///////////////////////////
    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&time, start, stop);
    // printf ("CUDA total time: %f ms\n", time);
    // ///////////////////////////

    hipFree(dev_input);
    hipFree(dev_polar_input);
    hipFree(dev_ccf_2d);
    hipFree(dev_ccf_angle);

    return EXIT_SUCCESS;
}

int CudaReprojectAndCorrelateArray(float* input_data, int num_images, size_t input_image_stride, size_t input_row_stride,
                                   float* output_data, size_t output_row_stride,
                                   int rows, int cols, int r_min, int r_max, int polar_angles,
                                   float center_y, float center_x, float cval)
{
    float *dev_input, *dev_polar_input, *dev_ccf_2d;
    float *dev_ccf_angle;
    size_t pitchInput, pitchPolar;
    
    // ///////////////////////////
    // hipEvent_t start, stop;
    // float time;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    // hipEventRecord(start, 0);
    // ///////////////////////////

    //Memory allocation for input and output arrays
    CUDA_CHECK(hipMallocPitch((void**)&dev_input, &pitchInput, sizeof(float)*cols, rows*num_images));
    CUDA_CHECK(hipMallocPitch((void**)&dev_polar_input, &pitchPolar, sizeof(float)*polar_angles, r_max-r_min));
    CUDA_CHECK(hipMallocPitch((void**)&dev_ccf_2d, &pitchPolar, sizeof(float)*polar_angles, r_max-r_min));
    CUDA_CHECK(hipMallocPitch((void**)&dev_ccf_angle, &pitchPolar, sizeof(float)*polar_angles, num_images));
    
    // Specify texture    
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    CUDA_CHECK(hipBindTexture2D(NULL, inputDataTexRef, dev_input, channelDesc, cols, rows*num_images, pitchInput));
    inputDataTexRef.addressMode[0] = hipAddressModeBorder;
    inputDataTexRef.addressMode[1] = hipAddressModeBorder;
    inputDataTexRef.filterMode = hipFilterModeLinear;
    inputDataTexRef.normalized = false;

    //Memory copying&initialisation for input data and error-check
    CUDA_CHECK(hipMemcpy2D(dev_input, pitchInput, input_data, input_row_stride, sizeof(float)*cols, rows*num_images, hipMemcpyHostToDevice));

    for(int n=0; n<num_images; n++){
        //Calculation    
        dim3 projBlock( 32, 32 );
        dim3 projGrid((r_max - r_min + projBlock.y - 1) / projBlock.y, 
                    (polar_angles + projBlock.x - 1) / projBlock.x);

        //Projection calculation
        gComputePolarProjection<<<projGrid,projBlock>>>(dev_polar_input, pitchPolar, n,
                                                        rows, cols, r_min, r_max, polar_angles,
                                                        center_y, center_x, 0);
        
        //RecoverMask
        dim3 recoverBlock( BLOCK_SIZE, 1 );
        dim3 recoverGrid( 1, r_max-r_min );
        gRecoverMask<<<recoverGrid,recoverBlock>>>( dev_polar_input, r_max-r_min, polar_angles, pitchPolar);

        dim3 corrBlock( BLOCK_SIZE, 1 );
        dim3 corrGrid( cols/BLOCK_SIZE + ((cols%BLOCK_SIZE==0)?0:1), rows );
        gCorrelationComputeLine<<<corrGrid,corrBlock>>>( dev_polar_input, dev_ccf_2d, r_max-r_min, polar_angles, pitchPolar);
        
        dim3 angleBlock( 1, BLOCK_SIZE );
        dim3 angleGrid( polar_angles, 1 );
        gCCFAngle<<<angleGrid,angleBlock>>>(dev_ccf_2d, (float *)((char *)dev_ccf_angle + n*pitchPolar), r_max-r_min, pitchPolar);
        CUDA_CHECK(hipDeviceSynchronize());

        CUDA_CHECK(hipMemcpy2D(output_data, output_row_stride, dev_ccf_angle, pitchPolar, polar_angles*sizeof(float), num_images, hipMemcpyDeviceToHost));
    }

    // ///////////////////////////
    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&time, start, stop);
    // printf ("CUDA total time: %f ms\n", time);
    // ///////////////////////////

    hipFree(dev_input);
    hipFree(dev_polar_input);
    hipFree(dev_ccf_2d);
    hipFree(dev_ccf_angle);

    return EXIT_SUCCESS;
}