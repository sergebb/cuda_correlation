#include "hip/hip_runtime.h"
#include "cuda_compute.h"
#include <stdio.h>
#include "hip/hip_math_constants.h"
 
#define BLOCK_SIZE 512
#define MAX_WIDTH 2048

texture<float, 2 > inputDataTexRef;

#define CUDA_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__
void gComputePolarProjection(float *dev_output, size_t pitchPolar,
                             int rows, int cols, int r_min, int r_max, int polar_angles,
                             float center_y, float center_x, float cval)
{
    unsigned int r = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int t = blockIdx.y * blockDim.y + threadIdx.y;
    
    float radius = r_min + r;
    float angle_step = 2*HIP_PI_F/polar_angles;
    float angle = -HIP_PI_F + t*angle_step ;
    float res;
    
    if(radius < r_max && t < polar_angles){
        float x = radius * cos(angle) + center_x + 1.0f;
        float y = radius * sin(angle) + center_y + 1.0f;
        if(x<0 || x>= cols || y<0 || y>= rows){
            res = cval;
        }else{
            res = tex2D(inputDataTexRef, x, y);
        }
        *((float*)((char*)dev_output + r * pitchPolar) + t) = res;
    }
}


__global__ void gCorrelationComputeLine( float *dev_input, float *dev_output, int rows, int cols, size_t pitchF) 
{
    int xCoord = blockIdx.x*blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    int yCoord = blockIdx.y;

    __shared__ float line_data[MAX_WIDTH];
    int i,pos;
    float val;
    float t;
    float sum = 0;
    if( xCoord<cols && yCoord<rows){
        for(i=0; i*BLOCK_SIZE < cols; i++){
            if(tid+i*BLOCK_SIZE < cols){
                line_data[tid + i*BLOCK_SIZE] = *((float*)((char*)dev_input + yCoord * pitchF) + tid + i*BLOCK_SIZE); //Copy line of matrix to shared memory
            }
        }
        __syncthreads();

        for( i=0; i<cols; i++ ){        //Loop over whole line
            val = line_data[i];         //First multiplier
            pos = (i+xCoord)%cols;      //Coordinate in shifted line(second line)
            t = line_data[pos];         //Value in memory cell (pos,yCoord), second multiplier
            sum += t * val;             //Sum of multiplications
        }
        sum /= cols;    //Divide by line len -> correlation
        
        *((float*)((char*)dev_output + yCoord * pitchF) + xCoord) = sum; //memory cell (xCoord,yCoord) where results of correlation is saved
    }
}
    
__global__ 
void gRecoverMask( float *dev_input, float *dev_mask, int rows, int cols, size_t pitchF) 
{
    int xCoord = threadIdx.x;
    int yCoord = blockIdx.y;
    int i,s;
    float val;
    float mask_val;
    float average_val;    
    
    __shared__ float sum_value;
    __shared__ int non_mask;
    __shared__ float sum_buf[BLOCK_SIZE];
    __shared__ float non_mask_buf[BLOCK_SIZE];
    
    if(xCoord == 0) {
        sum_value = 0;
        non_mask = 0;
    }
    for(i=0; i*BLOCK_SIZE < cols; i++){
        if(xCoord + i*BLOCK_SIZE < cols){
            val = *((float*)((char*)dev_input + yCoord * pitchF) + xCoord + i*BLOCK_SIZE);
            mask_val = *((float*)((char*)dev_mask + yCoord * pitchF) + xCoord + i*BLOCK_SIZE);
            non_mask = mask_val>=0 ? 1 : 0;
        }else{
            val = 0;
            non_mask = 0;
        }
        if(val < 0 || non_mask == 0){
            val = 0;
        }
        
        sum_buf[xCoord] = val;
        non_mask_buf[xCoord] = non_mask;
        for( s=1; s<BLOCK_SIZE; s*=2) {         //Sum reduction
            if(xCoord % (2*s) == 0) {
                sum_buf[xCoord] += sum_buf[xCoord + s];
                non_mask_buf[xCoord] += non_mask_buf[xCoord + s];
            }
            __syncthreads();
        }
        if(xCoord == 0) {
            sum_value += sum_buf[0];
            non_mask += non_mask_buf[0];
        }
    }
    __syncthreads();
    average_val = sum_value/non_mask;
    
    for(i=0; i*BLOCK_SIZE < cols; i++){
        if(xCoord + i*BLOCK_SIZE < cols){
            mask_val = *((float*)((char*)dev_mask + yCoord * pitchF) + xCoord + i*BLOCK_SIZE);
            if(mask_val < 0){
                *((float*)((char*)dev_input + yCoord * pitchF) + xCoord + i*BLOCK_SIZE) = average_val;
            }
        }
    }
}
__global__
void gCCFAngle(float *dev_ccf_2d, float *dev_ccf_angle, int radius_range, size_t pitchF){
    int xCoord = blockIdx.x;
    int yCoord = threadIdx.y;
    int i,s;
    __shared__ float sum_buf[BLOCK_SIZE];
    float sum_value = 0;
    
    for(i=0; i*BLOCK_SIZE < radius_range; i++){
        if(yCoord + i*BLOCK_SIZE < radius_range){
            sum_buf[yCoord] = *((float*)((char*)dev_ccf_2d + (yCoord + i*BLOCK_SIZE) * pitchF) + xCoord);
        }else{
            sum_buf[yCoord] = 0;
        }
        for( s=1; s<BLOCK_SIZE; s*=2) {         //Sum reduction
            if(yCoord % (2*s) == 0) {
                sum_buf[yCoord] += sum_buf[yCoord + s];
            }
            __syncthreads();
        }
        if(yCoord == 0){
            sum_value += sum_buf[0];
        }
    }
    if(yCoord == 0){
        dev_ccf_angle[xCoord] = sum_value/radius_range;
    }
    
}


int CudaReprojectToPolar(float **input_data, float **polar_data,
                         int rows, int cols, int r_min, int r_max, int polar_angles,
                         float center_y, float center_x, float cval)
{
    float *dev_input, *dev_output;
    float *t;
    size_t pitchInput, pitchPolar;
    
    // ///////////////////////
    // hipEvent_t start, stop;
    // float time;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    // hipEventRecord(start, 0);
    // ///////////////////////

    //Memory allocation for input and output arrays
    CUDA_CHECK(hipMallocPitch((void**)&dev_input, &pitchInput, sizeof(float)*cols, rows));
    CUDA_CHECK(hipMallocPitch((void**)&dev_output, &pitchPolar, sizeof(float)*polar_angles, r_max-r_min));

    //Memory copying&initialisation for input data and error-check
    for( int i=0; i<rows; i++ ){
        t = (float*)((char*)dev_input + i * pitchInput);
        CUDA_CHECK(hipMemcpy( t, input_data[i], pitchInput, hipMemcpyHostToDevice ));
    }
    for( int i=0; i<r_max-r_min; i++ ){
        t = (float*)((char*)dev_output + i * pitchPolar);
        CUDA_CHECK(hipMemset( t, 0, pitchPolar));
    }

    // Specify texture    
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    CUDA_CHECK(hipBindTexture2D(NULL, inputDataTexRef, dev_input, channelDesc, cols, rows, pitchInput));
    inputDataTexRef.addressMode[0] = hipAddressModeBorder;
    inputDataTexRef.addressMode[1] = hipAddressModeBorder;
    inputDataTexRef.filterMode = hipFilterModeLinear;
    inputDataTexRef.normalized = false;
    
    //Calculation    
    dim3 projBlock( 32, 32 );
    dim3 projGrid((r_max - r_min + projBlock.y - 1) / projBlock.y, 
                  (polar_angles + projBlock.x - 1) / projBlock.x);
    
    gComputePolarProjection<<<projGrid,projBlock>>>(dev_output, pitchPolar,
                                                    rows, cols, r_min, r_max, polar_angles,
                                                    center_y, center_x, cval);
    
    CUDA_CHECK(hipDeviceSynchronize());

    // Result memory copying back
    for( int i=0; i<r_max-r_min; i++ ){
        t = (float*)((char*)dev_output + i * pitchPolar);
        CUDA_CHECK(hipMemcpy( polar_data[i], t, polar_angles*sizeof(float), hipMemcpyDeviceToHost ));
    }
    
    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&time, start, stop);
    // printf ("CUDA execution time: %f ms\n", time);

    hipFree( dev_input );
    hipFree( dev_output );

    return EXIT_SUCCESS;
}

int CudaCorrelateLine(float** input_data, float** mask_data,float** output_data, int rows, int cols)
{
    float *dev_input, *dev_mask, *dev_output;
    size_t pitchF, pitchF2;
    
    if( cols > MAX_WIDTH ){
        fprintf( stderr, "Error at %s:%i : %s\n", __FILE__, __LINE__, "Image width exceeds max value, need recompile" );
        exit( -1 );
    } 
    
    //Memory allocation for input and output arrays
    CUDA_CHECK(hipMallocPitch((void**)&dev_input, &pitchF, sizeof(float)*cols, rows));
    CUDA_CHECK(hipMallocPitch((void**)&dev_mask, &pitchF2, sizeof(float)*cols, rows));
    CUDA_CHECK(hipMallocPitch((void**)&dev_output, &pitchF, sizeof(float)*cols, rows));
    
    //Memory copying&initialisation for input data and error-check
    for( int i=0; i<rows; i++ ){
        CUDA_CHECK(hipMemcpy(((char*)dev_input + i * pitchF), input_data[i], pitchF, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(((char*)dev_mask + i * pitchF), mask_data[i], pitchF, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemset(((char*)dev_output + i * pitchF), 0, pitchF));
    }
    
    //RecoverMask
    dim3 recoverBlock( BLOCK_SIZE, 1 );
    dim3 recoverGrid( 1, rows );
    gRecoverMask<<<recoverGrid,recoverBlock>>>( dev_input, dev_mask, rows, cols, pitchF);
    //Calculation
    dim3 corrBlock( BLOCK_SIZE, 1 );
    dim3 corrGrid( cols/BLOCK_SIZE + ((cols%BLOCK_SIZE==0)?0:1), rows );
    gCorrelationComputeLine<<<corrGrid,corrBlock>>>( dev_input, dev_output, rows, cols, pitchF);
    
    CUDA_CHECK(hipDeviceSynchronize());

    //Result memory copying back
    for( int i=0; i<rows; i++ ){
        CUDA_CHECK(hipMemcpy(output_data[i], ((char*)dev_output + i*pitchF), cols*sizeof(float), hipMemcpyDeviceToHost));
    }

    hipFree(dev_input);
    hipFree(dev_mask);
    hipFree(dev_output);
    
    return EXIT_SUCCESS;
}

int CudaReprojectAndCorrelate(float** input_data, float** mask_data,float* output_data, 
                              int rows, int cols, int r_min, int r_max, int polar_angles,
                              float center_y, float center_x, float cval)
{
    float *dev_input, *dev_mask, *dev_polar_input, *dev_polar_mask, *dev_ccf_2d;
    float *dev_ccf_angle;
    size_t pitchInput, pitchPolar;
    
    // ///////////////////////
    // hipEvent_t start, stop;
    // float time;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    // hipEventRecord(start, 0);
    // ///////////////////////

    //Memory allocation for input and output arrays
    CUDA_CHECK(hipMallocPitch((void**)&dev_input, &pitchInput, sizeof(float)*cols, rows));
    CUDA_CHECK(hipMallocPitch((void**)&dev_mask, &pitchInput, sizeof(float)*cols, rows));
    CUDA_CHECK(hipMallocPitch((void**)&dev_polar_input, &pitchPolar, sizeof(float)*polar_angles, r_max-r_min));
    CUDA_CHECK(hipMallocPitch((void**)&dev_polar_mask, &pitchPolar, sizeof(float)*polar_angles, r_max-r_min));
    CUDA_CHECK(hipMallocPitch((void**)&dev_ccf_2d, &pitchPolar, sizeof(float)*polar_angles, r_max-r_min));
    CUDA_CHECK(hipMalloc((void**)&dev_ccf_angle, sizeof(float)*polar_angles));

    //Memory copying&initialisation for input data and error-check
    for( int i=0; i<rows; i++ ){
        CUDA_CHECK(hipMemcpy( ((char*)dev_input + i * pitchInput), input_data[i], pitchInput, hipMemcpyHostToDevice ));
        CUDA_CHECK(hipMemcpy( ((char*)dev_mask + i * pitchInput), mask_data[i], pitchInput, hipMemcpyHostToDevice ));
    }
    for( int i=0; i<r_max-r_min; i++ ){
        CUDA_CHECK(hipMemset( ((char*)dev_polar_input + i * pitchPolar), 0, pitchPolar));
        CUDA_CHECK(hipMemset( ((char*)dev_polar_mask + i * pitchPolar), 0, pitchPolar));
        CUDA_CHECK(hipMemset( ((char*)dev_ccf_2d + i * pitchPolar), 0, pitchPolar));
    }

    //Calculation    
    dim3 projBlock( 32, 32 );
    dim3 projGrid((r_max - r_min + projBlock.y - 1) / projBlock.y, 
                  (polar_angles + projBlock.x - 1) / projBlock.x);

    // Specify texture    
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    CUDA_CHECK(hipBindTexture2D(NULL, inputDataTexRef, dev_input, channelDesc, cols, rows, pitchInput));
    inputDataTexRef.addressMode[0] = hipAddressModeBorder;
    inputDataTexRef.addressMode[1] = hipAddressModeBorder;
    inputDataTexRef.filterMode = hipFilterModeLinear;
    inputDataTexRef.normalized = false;
    
    //Projection calculation
    gComputePolarProjection<<<projGrid,projBlock>>>(dev_polar_input, pitchPolar,
                                                    rows, cols, r_min, r_max, polar_angles,
                                                    center_y, center_x, 0);
    
    CUDA_CHECK(hipBindTexture2D(NULL, inputDataTexRef, dev_mask, channelDesc, cols, rows, pitchInput));
    inputDataTexRef.addressMode[0] = hipAddressModeBorder;
    inputDataTexRef.addressMode[1] = hipAddressModeBorder;
    inputDataTexRef.filterMode = hipFilterModeLinear;
    inputDataTexRef.normalized = false;
    
    gComputePolarProjection<<<projGrid,projBlock>>>(dev_polar_mask, pitchPolar,
                                                    rows, cols, r_min, r_max, polar_angles,
                                                    center_y, center_x, -10000);
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipUnbindTexture(inputDataTexRef));
    
    //RecoverMask
    dim3 recoverBlock( BLOCK_SIZE, 1 );
    dim3 recoverGrid( 1, r_max-r_min );
    gRecoverMask<<<recoverGrid,recoverBlock>>>( dev_polar_input, dev_polar_mask, r_max-r_min, polar_angles, pitchPolar);

    dim3 corrBlock( BLOCK_SIZE, 1 );
    dim3 corrGrid( cols/BLOCK_SIZE + ((cols%BLOCK_SIZE==0)?0:1), rows );
    gCorrelationComputeLine<<<corrGrid,corrBlock>>>( dev_polar_input, dev_ccf_2d, r_max-r_min, polar_angles, pitchPolar);
    
    dim3 angleBlock( 1, BLOCK_SIZE );
    dim3 angleGrid( polar_angles, 1 );
    gCCFAngle<<<angleGrid,angleBlock>>>(dev_ccf_2d, dev_ccf_angle, r_max-r_min, pitchPolar);
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy( output_data, dev_ccf_angle, polar_angles*sizeof(float), hipMemcpyDeviceToHost ));

    // ///////////////////////////
    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&time, start, stop);
    // printf ("CUDA total time: %f ms\n", time);
    // ///////////////////////////

    hipFree(dev_input);
    hipFree(dev_mask);
    hipFree(dev_polar_input);
    hipFree(dev_polar_mask);
    hipFree(dev_ccf_2d);
    hipFree(dev_ccf_angle);

    return EXIT_SUCCESS;
}